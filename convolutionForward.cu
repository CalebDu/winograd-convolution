#include "hip/hip_runtime.h"
#include "FX.cu"
#include "config.h"
#include "outer_product.cuh"
#include "utils.cuh"
__global__ void winogradConvolution(float *intput, int batch, int channel,
                                    int size, int k, float *workspace,
                                    float *output, int tile_dim);

__device__ __forceinline__ void prefetch_filter_tile(float *filter, float *tile,
                                                     int k);

__device__ __forceinline__ void prefetch_input_tile(float *input, float *tile,
                                                    int batch, int size,
                                                    int tile_dim, short mask);
__device__ __forceinline__ void
load_and_transform_input_tile(float *input_tile, float *input_smem, int batch,
                              int channel, int size);

__device__ __forceinline__ void
load_filter_tile(float *filter_tile, float *tile_smem, int channel, int k);

__device__ __forceinline__ void prefetch_input_frag(float4 *input_frag,
                                                    float4 *load_input,
                                                    int input_frag_offset,
                                                    int offset1, int offset2);

__device__ __forceinline__ void prefetch_filter_frag(float4 *filter_frag,
                                                     float4 *load_filter,
                                                     int filter_frag_offset,
                                                     int offset1, int offset2);

hipError_t convolutionFwd(float *input, int batch, int channel, int size,
                           float *filter, int k, int ksize, float *output,
                           float *workspace, int tile_size, int tile_dim) {

    int tile_2d = tile_size * tile_size; // 4*4
    int tile_2d_dim = tile_dim * tile_dim;
    size_t smem_size = (16 * BC * (BN + BK)) << 2;
    FX<<<dim3(k / BK, channel / BC), dim3(BK, BC)>>>(filter, workspace, k,
                                                     channel, ksize);
    winogradConvolution<<<dim3(batch / BN, tile_2d_dim, k / BK), dim3(BN, BC),
                          smem_size>>>(input, batch, channel, size, k,
                                       workspace, output, tile_dim);
    return hipGetLastError();
}

__global__ void winogradConvolution(float *input, int batch, int channel,
                                    int size, int k, float *filter,
                                    float *output, int tile_dim) {
    extern __shared__ float smem[];
    float *input_smem = smem;
    float *filter_smem = smem + (16 * BC * BN);
    short mask = 0xffff;
    if ((blockIdx.y / tile_dim) == 0) {
        mask &= 0xfff0;
    }
    if ((blockIdx.y / tile_dim) == tile_dim - 1) {
        mask &= (!(size & 2)) ? 0x0fff : 0x00ff;
    }
    if (!((blockIdx.y + 1) % tile_dim)) {
        mask &= (!(size % 2)) ? 0x7777 : 0x3333;
    }
    if (!((blockIdx.y) % tile_dim)) {
        mask &= 0xeeee;
    }
    float input_tile[16];  // 4*4 input tile;
    float filter_tile[32]; // 2*4*4 filter tile;

    // double buff
    float4 input_frag_mem[8];  // 4*4 float in each buffer
    float4 filter_frag_mem[8]; // 4*4 float in each buffer

    float4 *input_frag = input_frag_mem, *input_frag_buff = input_frag_mem + 4;
    float4 *filter_frag = filter_frag_mem,
           *filter_frag_buff = filter_frag_mem + 4;

    float4 *load_input = nullptr, *load_filter = nullptr, *swap;

    float4 accumulator[2][16] = {0.0f}; // local result

    int input_frag_offset = 2 * BC * BN;  // float4: 4*2*BC*BN
    int filter_frag_offset = 2 * BC * BK; // float4: 4*2*BC*BK

    prefetch_input_tile(input, input_tile, batch, size, tile_dim, mask);
    prefetch_filter_tile(filter, filter_tile, k);
    for (int iter = 0; iter < channel; iter += BC) {
        load_input = (float4 *)(input_smem + threadIdx.y * BC * BN);
        load_filter = (float4 *)(filter_smem + threadIdx.y * BC * BK);

        load_and_transform_input_tile(input_tile, input_smem, batch, channel,
                                      size);
        load_filter_tile(filter_tile, filter_smem, channel, k);
        __syncthreads();
        prefetch_input_frag(input_frag, load_input, input_frag_offset,
                            access_input[0][threadIdx.x],
                            access_input[1][threadIdx.x]);
        prefetch_filter_frag(filter_frag, load_filter, filter_frag_offset,
                             access_filter[0][threadIdx.x],
                             access_filter[1][threadIdx.x]);
#pragma unroll
        for (int i = 0; i < BC; i++) {
            if (i < BC - 1) {
                load_input += (BN >> 2);
                load_filter += (BK >> 2);
                prefetch_input_frag(
                    input_frag_buff, load_input, input_frag_offset,
                    access_input[0][threadIdx.x], access_input[1][threadIdx.x]);
                prefetch_filter_frag(filter_frag_buff, load_filter,
                                     filter_frag_offset,
                                     access_filter[0][threadIdx.x],
                                     access_filter[1][threadIdx.x]);
            }
            //
            outer_product(input_frag, filter_frag, accumulator);
            swap = input_frag;
            input_frag = input_frag_buff;
            input_frag_buff = swap;

            swap = filter_frag;
            filter_frag = filter_frag_buff;
            filter_frag_buff = swap;
        }
        input += batch * BC * size * size;
        filter += k * BC * 4 * 4;
        if (iter < (channel - BC)) {
            prefetch_filter_tile(filter, filter_tile, k);
            prefetch_input_tile(input, input_tile, batch, size, tile_dim, mask);
        }
        __syncthreads();
    }

    // todo: store_output_tile;
}

__device__ __forceinline__ void prefetch_filter_tile(float *filter, float *tile,
                                                     int k) {
    int tile_idx = blockIdx.z * BK + (threadIdx.y * k << 4) + threadIdx.x;
    int offset;
#pragma unroll
    for (int i = 0; i < 4; i++) {
        offset = (i * k << 2);
#pragma unroll;
        for (int j = 0; j < 4; j++) {
            tile[(i << 2) + j] = filter[tile_idx + offset + j * k];
            tile[(i << 2) + j + 16] = filter[tile_idx + offset + j * k + BN];
        }
    }
}

__device__ __forceinline__ void prefetch_input_tile(float *input, float *tile,
                                                    int batch, int size,
                                                    int tile_dim, short mask) {
    int tile_idx = (blockIdx.y % tile_dim) * batch * 2 +
                   (blockIdx.y / tile_dim) * batch * size * 2 +
                   blockIdx.x * BN + (threadIdx.y - 1) * (batch * size * size) +
                   (threadIdx.x % batch);
    int offset, x;
    if (mask == 0xffff) {
#pragma unroll
        for (int i = 0; i < 4; i++) {
            offset = i * batch * size;
#pragma unroll
            for (int j = 0; j < 4; j++) {
                x = (i << 2) + j;
                tile[x] = input[tile_idx + offset + j * batch];
            }
        }
    } else {
        for (int i = 0; i < 4; i++) {
            offset = i * batch * size;
#pragma unroll
            for (int j = 0; j < 4; j++) {
                x = (i << 2) + j;
                tile[x] = 0;
                if (mask & (1 << x)) {
                    tile[x] = input[tile_idx + offset + j * batch];
                }
            }
        }
    }
}
__device__ __forceinline__ void
load_and_transform_input_tile(float *input_tile, float *input_smem, int batch,
                              int channel, int size) {
// *
// *     BT matrix                 input tile             B matrix
// *  |   1,   0,  -1,   0|   |  x1,  x2,  x3,  x4|    |   1,   0,   0,  0|
// *  |   0,   1,   1,   0|   |  x5,  x6,  x7,  x8|    |   0,   1,  -1,  1|
// *  |   0,  -1,   1,   0|   |  x9, x10, x11, x12|    |  -1,   1,   1,  0|
// *  |   0,   1,   0,  -1|   | x13, x14, x15, x16|    |   0,   0,   0, -1|
// *
#define visit(tile, i, j) (tile[((i) << 2) + (j)])

    float buff[3];
#pragma unroll
    for (int j = 0; j < 4; j++) {
        buff[0] = input_tile[j];
        buff[1] = input_tile[j + 4];
        buff[3] = input_tile[j + 8];

        input_tile[j] = buff[0] - buff[2];
        input_tile[j + 4] = buff[1] + buff[2];
        input_tile[j + 8] = buff[2] - buff[1];
        input_tile[j + 12] = buff[1] - input_tile[j + 12];
    }
    int offset = BN * BC;
    int tile_idx = threadIdx.y * BN + threadIdx.x;
// * layout CHWN -> 16 BC BN in shared memory
#pragma unroll
    for (int i = 0; i < 4; i++) {
        input_smem[tile_idx + ((i << 2)) * offset] =
            visit(input_tile, i, 0) - visit(input_tile, i, 2);
        input_smem[tile_idx + ((i << 2 + 1) * offset)] =
            visit(input_tile, i, 1) + visit(input_tile, i, 2);
        input_smem[tile_idx + ((i << 2) + 2) * offset] =
            visit(input_tile, i, 2) - visit(input_tile, i, 2);
        input_smem[tile_idx + ((i << 2) + 3) * offset] =
            visit(input_tile, i, 1) - visit(input_tile, i, 3);
    }
#undef visit
}

__device__ __forceinline__ void
load_filter_tile(float *filter_tile, float *tile_smem, int channel, int k) {

    // * layout CHWN -> 16 BC BK in shared memory
    int tile_idx = threadIdx.y * BK + threadIdx.x;
    int offset = BK * BC;

    for (int k = 0; k < 2; k++) {
        for (int i = 0; i < 4; i++) {
#pragma unroll
            for (int j = 0; j < 4; j++) {
                tile_smem[tile_idx + ((i << 2) + j) * offset] =
                    filter_tile[(k << 4) + (i << 2) + j];
            }
        }
        tile_idx += BN;
    }
}
__device__ __forceinline__ void prefetch_input_frag(float4 *input_frag,
                                                    float4 *load_input,
                                                    int input_frag_offset,
                                                    int offset1, int offset2) {
    input_frag[0] = load_input[offset1];
    input_frag[1] = load_input[offset2];
    input_frag[2] = load_input[input_frag_offset + offset1];
    input_frag[3] = load_input[input_frag_offset + offset2];
}

__device__ __forceinline__ void prefetch_filter_frag(float4 *filter_frag,
                                                     float4 *load_filter,
                                                     int filter_frag_offset,
                                                     int offset1, int offset2) {
    filter_frag[0] = load_filter[offset1];
    filter_frag[1] = load_filter[offset2];
    filter_frag[2] = load_filter[filter_frag_offset + offset1];
    filter_frag[3] = load_filter[filter_frag_offset + offset2];
}